
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <iostream>


#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)


// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) 
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	// Use atomicAdd
	int sum = A[idx] + B[idx];
 	atomicAdd(C, sum);

}


int main() 
{
	
	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
	}

	*h_C = 0;


	// Allocate device memory 
	hipMalloc((void**)&d_A, DSIZE * sizeof(int));
	hipMalloc((void**)&d_B, DSIZE * sizeof(int));
	hipMalloc((void**)&d_C, sizeof(int));
	
	// Check memory allocation for errors
	cudaCheckErrors();

	// Copy the matrices on GPU
	hipMemcpy(d_A, h_A, DSIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, DSIZE * sizeof(int), hipMemcpyHostToDevice);
	
	// Check memory copy for errors
	cudaCheckErrors();

	// Define block/grid dimentions and launch kernel
	const int block_size = 256;
	const int grid_size = DSIZE/block_size;
	dot_product<<<grid_size, block_size>>>(d_A, d_B, d_C, DSIZE);
	
	// Copy results back to host
	hipMemcpy(h_C, d_C, sizeof(int), hipMemcpyDeviceToHost);
	
    // Check copy for errors
	cudaCheckErrors();

	// Verify result
	std::cout << "A • B = " << *h_C << std::endl;

	// Free allocated memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	delete[] h_A;
	delete[] h_B;
	delete h_C;
	return 0;

}